
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main()
{
    int a = 100;
    int b = 200;
    int c = -1;

    int *device_ptr_c;

    hipError_t error = hipMalloc(&device_ptr_c, sizeof(int));
    if (error != hipSuccess) {
        std::cerr << "cuda malloc failed, " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }

    add<<<1,1>>>(a, b, device_ptr_c);

    error = hipMemcpy(&c, device_ptr_c, sizeof(int), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        std::cerr << "cuda memcpy failed, " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
    
    std::cout << a << " + " << b << " = " << c << std::endl;

    hipFree(device_ptr_c);

    return 0;
}
