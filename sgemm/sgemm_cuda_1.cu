#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <string>
#include "cuda_utils.h"
#include "data.h"

template<int BLOCK>
__global__ void sgemm(int m, int k, int n, const float *x, const float *y, float *z) {
    int mi = blockIdx.x * BLOCK + threadIdx.x;
    int ni = blockIdx.y * BLOCK + threadIdx.y;

    if ((mi < m) && (ni < n)) {
        double out = 0.0f;
        for (int s = 0; s < k; s++) {
            out += x[mi * k + s] * y[s * n + ni];
        }
        z[mi * n + ni] = out; 
    }
}

int main() {
    constexpr int kDim = 1024;
    int m = kDim;
    int k = kDim;
    int n = kDim;

    size_t size_x = sizeof(float) * m * k;
    size_t size_y = sizeof(float) * k * n;
    size_t size_z = sizeof(float) * m * n;

    // 设置host侧数据
    std::string bin_prefix = kDataDir + "/sgemm_m_" + std::to_string(m) + "_k_" + std::to_string(k) + "_n_" + std::to_string(n);
    float * host_ptr_x = read_bin(bin_prefix +"_x.bin", size_x);
    float * host_ptr_y = read_bin(bin_prefix +"_y.bin", size_y);
    float * expected_z = read_bin(bin_prefix +"_z.bin", size_z);
    float * host_ptr_z = (float *)malloc(size_z);

    // 设置device侧数据
    float *device_ptr_x;
    float *device_ptr_y;
    float *device_ptr_z;

    HANDLE_ERROR(hipMalloc(&device_ptr_x, size_x));
    HANDLE_ERROR(hipMalloc(&device_ptr_y, size_y));
    HANDLE_ERROR(hipMalloc(&device_ptr_z, size_z));

    HANDLE_ERROR(hipMemcpy(device_ptr_x, host_ptr_x, size_x, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(device_ptr_y, host_ptr_y, size_y, hipMemcpyHostToDevice));
    
    // 计算matmul
    constexpr int kBLOCK = 16;
    dim3 block(kBLOCK, kBLOCK);
    dim3 grid(CEIL_DIV(m, kBLOCK), CEIL_DIV(n, kBLOCK));
    sgemm<kBLOCK><<<grid, block>>>(m, k, n, device_ptr_x, device_ptr_y, device_ptr_z);
    HANDLE_ERROR(hipMemcpy(host_ptr_z, device_ptr_z, size_z, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(device_ptr_x));
    HANDLE_ERROR(hipFree(device_ptr_y));
    HANDLE_ERROR(hipFree(device_ptr_z));

    // 比较计算结果
    printf("calculated z:\n");
    print_array(host_ptr_z, m * n);
    printf("expected z:\n");
    print_array(expected_z, m * n);

    bool is_equal = allclose(host_ptr_z, expected_z, m * n);
    printf("\nAccuracy checking result: %s.\n", (is_equal ? "PASS" : "NOT PASS!!!"));

    free(host_ptr_x);
    free(host_ptr_y);
    free(host_ptr_z);
    free(expected_z);
}